/*
 * This is an introductory exercise in using constant memory
 * and then shared memory. The reserve array idea is from an
*  NVIDIA exercise of the same type.
 *
 * Training material developed by Kevin Stratford
 * Copyright EPCC, The University of Edinburgh, 2017 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Forward Declaration*/
/* Utility function to check for and report CUDA errors */

void checkCUDAError(const char*);

/*
 * The number of CUDA threads per block to use.
 */

#define THREADS_PER_BLOCK 128

/* The number of integer elements in the array */

static __constant__ int constArraySize;

/* Reverse the elements in the input array d_in.
 * The total number of threads should be ARRAY_SIZE. */

__global__ void reverseArray(int * d_in, int * d_out)
{
  __shared__ int sh[THREADS_PER_BLOCK];
  int read_idx = blockIdx.x*blockDim.x + threadIdx.x;
  int  tmp_idx = THREADS_PER_BLOCK - (threadIdx.x + 1);
  sh[tmp_idx] = d_in[read_idx];
  __syncthreads();

  int num_blocks = constArraySize / THREADS_PER_BLOCK;
  int write_block = num_blocks - (blockIdx.x + 1);
  int write_idx = write_block * blockDim.x + threadIdx.x;
  d_out[write_idx] = sh[threadIdx.x];
}


/* Main routine */
void TestArray(int ARRAY_SIZE)
{
    printf("\n\nTesting array size %d\n\n", ARRAY_SIZE);
    hipMemcpyToSymbol(HIP_SYMBOL(constArraySize), &ARRAY_SIZE, sizeof(ARRAY_SIZE)/*,
			0, cudaMemcpyHostToDevice*/);
    checkCUDAError("Copying constant ARRAY_SIZE to symbol table");

    int *h_in, *h_out;
    int *d_in, *d_out;

    int i;
    int ncorrect;
    const size_t sz = ARRAY_SIZE * sizeof(int);

    /*
     * allocate memory on host
     * h_in holds the input array, h_out holds the result
     */
    h_in = (int *) malloc(sz);
    h_out = (int *) malloc(sz);

    /*
     * allocate memory on device
     */
    hipMalloc(&d_in, sz);
    hipMalloc(&d_out, sz);

    /* initialise host arrays */
    for (i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = i;
        h_out[i] = 0;
    }

    /* copy input array from host to GPU */

    hipMemcpy(d_in, h_in, sz, hipMemcpyHostToDevice);

    /* run the kernel on the GPU */

    dim3 blocksPerGrid(ARRAY_SIZE/THREADS_PER_BLOCK, 1, 1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK, 1, 1);

    reverseArray<<< blocksPerGrid, threadsPerBlock >>>(d_in, d_out);

    /* wait for all threads to complete and check for errors */

    hipDeviceSynchronize();
    checkCUDAError("kernel invocation");

    /* copy the result array back to the host */

    hipMemcpy(h_out, d_out, sz, hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpyDeviceToHost");

    /* print out the result */
    printf("Results: ");
    ncorrect = 0;
    for (i = 0; i < ARRAY_SIZE; i++) {
      if (h_out[i] == h_in[ARRAY_SIZE - (i+1)]) ncorrect += 1;
    }
    printf("Number of correctly reversed elements %d (%s)\n", ncorrect,
           ncorrect == ARRAY_SIZE ? "Correct" : "INCORRECT");
    printf("\n");

    /* free device buffers */

    hipFree(d_out);
    hipFree(d_in);

    /* free host buffers */
    free(h_in);
    free(h_out);
}

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        printf("Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

/* Main routine */
int main(int argc, char *argv[])
{
  /* Print device details */
  int deviceNum;
  hipGetDevice(&deviceNum);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, deviceNum);
  printf("  Device name: %s\n", prop.name);

  TestArray(65536);
  TestArray(0x20000);
}
