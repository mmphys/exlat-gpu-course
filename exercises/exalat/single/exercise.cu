/*
 * Skeleton for Very Basic Linear Solver Development
 *
 * README
 * This template should compile, but is missing relevant functionality.
 * The exercise is to add, step-by-step, the necessary code.
 * STEPS are indicated by comments in the code, e.g.,
 *
 * STEP 1.1(a) Create a vector dot product kernel.
 *
 * Nick Johnson, EPCC && ExaLAT.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char*);

/* The number of integer elements in the array */
#define ARRAY_SIZE 32

/*
 * The number of CUDA blocks and threads per block to use.
 * These should always multiply to give the array size.
 * For the single block kernel, NUM_BLOCKS should be 1 and
 * THREADS_PER_BLOCK should be the array size
 */
#define THREADS_PER_BLOCK 32
/*#define NUM_BLOCKS 32*/

/* Define max number of devices we expect per node.
 * It's currently 8 on Cirrus, so we keep to that for now. */

/*#define MAX_DEVICES 8*/


/*
 * Vector Vector product (dot product)
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * output: result, pointer to a previously allocated scalar which will
 * contain the dot product A.B
 */

__global__ void vector_vector(float *vectorA, float *vectorB, float *result) {

  /*
   * STEP 1.1(a) Implement your vector dot product here.
   * STEP 1.1(b) Invoke the kernel from the main loop.
   */
  __shared__ float sum;
  __shared__ float sh[THREADS_PER_BLOCK];
  sum = 0;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  sh[threadIdx.x] = vectorA[idx] * vectorB[idx];
  __syncthreads();
  atomicAdd(&sum, sh[threadIdx.x]);
  *result = sum;
}

void check_vector_vector(float *vectorA, float *vectorB, float *result)
{
  float check = 0;
  for( int i = 0; i < ARRAY_SIZE; ++i )
    check += vectorA[i] * vectorB[i];
  if( check != *result )
  {
    printf( "check_vector_vector failed\n" );
    exit(EXIT_FAILURE);
  }
  printf( "check_vector_vector success\n" );
}

/*
 * Matrix Vector product
 * input:  matrix, pointer to matrix with flattened 1-d addressing
 *         A_ij = matrix[j*ARRAY_SIZE + i]
 * input:  vector, pointer to a previously allocated vector
 * output: result, pointer to a previously allocated vector
 */
 
__global__ void matrix_vector(float *matrix, float *vector, float *result) {


  /* STEP 1.2(a) Implement you matrix vector product here.
   * STEP 1.2(b) Invoke the kernel from the main code below and
   * check the result. */
  
  __shared__ float sum;
  __shared__ float sh[THREADS_PER_BLOCK];
  sum = 0;
  int idxM = blockIdx.x * blockDim.x + threadIdx.x;
  int idxV = threadIdx.x;
  sh[idxV] = matrix[idxM] * vector[idxV];
  __syncthreads();
  atomicAdd(&sum, sh[idxV]);
  result[blockIdx.x] = sum;
}

void check_matrix_vector(float *matrix, float *vector, float *result)
{
  float check[ARRAY_SIZE];
  for( int i = 0; i < ARRAY_SIZE; ++i )

  for( int i = 0; i < ARRAY_SIZE; ++i )
  {
    check[i] = 0;
    for( int j = 0; j < ARRAY_SIZE; ++j )
      check[i] += matrix[i * ARRAY_SIZE + j] * vector[j];
    if( check[i] != result[i] )
    {
      printf( "check_matrix_vector failed on row %d\n", i );
      exit(EXIT_FAILURE);
    }
  }
  printf( "check_vector_vector success\n" );
}

/*
 * Vector plus Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * output: resvector, pointer to a previously allocated vector
 * which will contain the elementwise sum a_i + b_i.
 */
 
__global__ void vector_add(float *vectorA, float *vectorB, float *resvector) {

  /* STEP 1.3(a) Implement your vector addition here */
  /* STEP 1.3(b) Implement the kernel launch in the main code and check
   * your result for known input */

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  resvector[idx] = vectorA[idx] + vectorB[idx];

}

/*
 * Vector plus Factor * Vector
 * input:  vectorA, pointer to a previously allocated vector
 * input:  vectorB, pointer to a previously allocated vector
 * input:  factor, a scalar which elementwise multiplies the second vector
 * output: resscalar, pointer to a previously allocated vector which
 * contain the result elementwise a_i + f b_i
 */
 
__global__ void vector_add_factor(float *vectorA, float *vectorB, float factor, float *resvector) {

  /* STEP 1.4(a) implement the kernel here, and check the kernel
   * invocation in the main code. */

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  resvector[idx] = vectorA[idx] + factor * vectorB[idx];
}

void check_vector_add_factor(float *vectorA, float *vectorB, float factor, float *resvector)
{
  for( int i = 0; i < ARRAY_SIZE; ++i )
  {
    if( resvector[i] != vectorA[i] + vectorB[i] * factor )
    {
      printf( "check_vector_add_factor failed element %d\n", i );
      exit(EXIT_FAILURE);
    }
  }
  printf( "check_vector_add_factor success\n" );
}

/*
 * Function which seeds a square matrix of ARRAY_SIZE x ARRAY_SIZE
 * with positive values on the diagonal.
 */

__host__ int seedmatrix(float *matrix) {

  int i = 0;
  int j = 0;

  for (j = 0; j < ARRAY_SIZE; j++) {
    for (i = 0; i< ARRAY_SIZE; i++) {
      if (i == j) {
        matrix[j*ARRAY_SIZE +i] = 1.0;
      }
      else{
        matrix[j*ARRAY_SIZE +i] = 0.0;
      }

    }
  }

  return 0;
}



/* Main function */

int main(int argc, char *argv[]) {

  /*
   * This is pre-amble code to deal with multiple GPUs, please do not edit.
   */
  
  /*
   * Check that there are some GPUs, but not too many
   */

   int cuda_device_count = 0;
   hipGetDeviceCount(&cuda_device_count);

   if (cuda_device_count == 0) {
     printf("No GPU devices found!\n");
     return -1;
   }

  /*
   * We print out the properties of each CUDA device for information.
   */

  int i = 0;
  hipDeviceProp_t prop;

  printf("Number of CUDA Devices = %d\n", cuda_device_count);
  for (i = 0; i < cuda_device_count; i++) {
    hipGetDeviceProperties(&prop, i);
    printf("\tDevice %d : Device name: %s\n", i, prop.name);
  }
  printf("\n");

  /*
   * End pre-amble
   */

  cuda_device_count = 1;
  printf("Set cuda_device_count to 1\n");


  /*
   * Begin main code
   */


  /*
   * Some useful helper sizes and variables
   */
  /*int j = 0;*/
  size_t matrix_sz = ARRAY_SIZE * ARRAY_SIZE * sizeof(float);
  size_t vector_sz = ARRAY_SIZE * sizeof(float);
  size_t scalar_sz = 1 * sizeof(float);
  float scalar = 0;

  

  /*
   * Create pointers to hold data on the host
   */
  float *matrixA = NULL;
  float *vectorR = NULL;
  float *vectorB = NULL;
  float *vectorX = NULL;
  float *vectorP = NULL;

  float *vectorRnew = NULL;
  float *vectorXnew = NULL;
  float *vectorPnew = NULL;

  /*
   * Allocate memory on host & test it was successful
   * This is an often missed step and can catch you out
   * We use heap allocations rather than stack for two reasons
   * 1. It makes everything a pointer which marries nicely with cudaMalloc
   * 2. It would be easy to fill the stack space and we cannot use ulimit on all systems to increase it
   */
  matrixA = (float *) calloc(matrix_sz, 1);
  vectorR = (float *) calloc(vector_sz, 1);
  vectorB = (float *) calloc(vector_sz, 1);
  vectorP = (float *) calloc(vector_sz, 1);
  vectorX = (float *) calloc(vector_sz, 1);
  vectorRnew = (float *) calloc(vector_sz, 1);
  vectorXnew = (float *) calloc(vector_sz, 1);
  vectorPnew = (float *) calloc(vector_sz, 1);
  
  if (matrixA == NULL ||\
      vectorR == NULL ||\
      vectorB == NULL ||\
      vectorP == NULL ||\
      vectorX == NULL ||\
      vectorRnew == NULL ||\
      vectorXnew == NULL ||\
      vectorPnew == NULL){
    printf("Error allocating host memory.\n");
    return 1;
  }

  /*
   * Initialise host arrays
   * Calloc should push these to be 0, but using this method we can pick anything.
   * Having a non-zero initialiser for the output array can help spot problems if we never expect a 0 in the output
   */
  seedmatrix(matrixA);
  for (int j = 0; j < ARRAY_SIZE; j++){
    vectorP[j] = 0.0;
    vectorB[j] = 1.0;
    vectorX[j] = 2.0;
    vectorRnew[j] = 0.0;
    vectorXnew[j] = 0.0;
    vectorPnew[j] = 0.0; 
  }



  
  /*
   * Create pointers to hold data on the device
   */
  float *device_matrixA = NULL;
  float *device_vectorR = NULL;
  float *device_vectorB = NULL;
  float *device_vectorX = NULL;
  float *device_vectorP = NULL;
  
  float *device_vectorRnew = NULL;
  float *device_vectorXnew = NULL;
  float *device_vectorPnew = NULL;
  float *device_vectorAP = NULL;
    
  float *device_scalar = NULL;



  /*
   * Set up some useful values
   * threadsPerBlock is as in previous exercises and simply a cast from the macro define
   * nBlocks is split across the number of devices we have
   */
  dim3 threadsPerBlock(THREADS_PER_BLOCK);
  /*dim3 nBlocks(NUM_BLOCKS/cuda_device_count);*/
  dim3 vBlocks(ARRAY_SIZE / THREADS_PER_BLOCK);
  dim3 mBlocks(ARRAY_SIZE * ARRAY_SIZE / THREADS_PER_BLOCK);

  printf("vBlocks: %d\n", vBlocks.x);
  printf("mBlocks: %d\n", vBlocks.x);
  printf("threadsPerBlock: %d\n", threadsPerBlock.x);

  /*
   * The compiler ignores pragmas statements which it cannot parse, so this can live outside the guard
   * The value of cuda_k will be 0 for a serial case so we get a single iteration of this loop
   * and ergo a single thread of execution.
   */

  for (int cuda_device = 0; cuda_device < cuda_device_count; cuda_device++) {
   
    hipSetDevice(cuda_device);
    hipGetDeviceProperties(&prop, cuda_device);
    
    printf("Allocate for device: %d %s\n", cuda_device, prop.name);
  
    /*
     * Allocate device memory
     * This is done inside the loop to give us some flexibility in a multi-GPU case
     */
    hipMalloc(&device_matrixA, matrix_sz/cuda_device_count);
    checkCUDAError("Device matrixA allocation");
    hipMalloc(&device_vectorR, vector_sz);
    checkCUDAError("Device vectorR allocation");
    hipMalloc(&device_vectorB, vector_sz);
    checkCUDAError("Device vectorB allocation");
    hipMalloc(&device_vectorP, vector_sz);
    checkCUDAError("Device vectorP allocation");
    hipMalloc(&device_vectorX, vector_sz);
    checkCUDAError("Device vectorX allocation");

    hipMalloc(&device_vectorRnew, vector_sz);
    checkCUDAError("Device vectorRnew allocation");
    hipMalloc(&device_vectorPnew, vector_sz);
    checkCUDAError("Device vectorPnew allocation");
    hipMalloc(&device_vectorXnew, vector_sz);
    checkCUDAError("Device vectorXnew allocation");

    hipMalloc(&device_vectorAP, vector_sz);
    checkCUDAError("Device vectorAP allocation");

    hipMalloc(&device_scalar, scalar_sz);
    checkCUDAError("Device vectorXnew allocation");



    /*
     * This is the start of the initialisation step
     * We must compute an initial residual r_0, and set p = r_0
     */

    
    /*
     * Copy arrays and matrices to device(s)
     * The offset arrangement helps with >1 GPU
     */
    hipMemcpy(device_matrixA, matrixA, matrix_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D matrix");
    hipMemcpy(device_vectorX, vectorX, vector_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D vectorX");

    /*
     * Compute Ax_0 and keep the result vector in device memory
     */

    /* STEP 1.2(b) Use matrix_vector<<<>>>(); */

  matrix_vector<<<mBlocks,threadsPerBlock>>>(device_matrixA, device_vectorX, device_vectorXnew);
  hipMemcpy(vectorXnew, device_vectorXnew, vector_sz, hipMemcpyDeviceToHost);
  checkCUDAError("kernel invocation");

  check_matrix_vector(matrixA, vectorX, vectorXnew);

    /*
     * Compute the initial residual r_0 = b - (Ax_0) 
     */
    hipMemcpy(device_vectorB, vectorB, vector_sz, hipMemcpyHostToDevice);
    vector_add_factor<<<vBlocks, threadsPerBlock>>>(device_vectorB, device_vectorXnew, -1., device_vectorR);

    /* STEP 1.4(b) use (with f = -1.0)  vector_add_factor<<<>>>(); */

    /*
     * Copy the initial residual vector back to the host
     */
    hipMemcpy(vectorR, device_vectorR, vector_sz, hipMemcpyDeviceToHost);
    check_vector_add_factor(vectorB, vectorXnew, -1., vectorR);

    /* Set p_0 = r_0, copy this initial r to device p host side only! */
    memcpy(vectorP, vectorR, vector_sz);


    scalar = 0;
    hipMemcpy(device_scalar, &scalar, scalar_sz, hipMemcpyHostToDevice);
    checkCUDAError("Memcpy: H2D scalar");
    hipDeviceSynchronize();

    /*
     * Compute r_0 r_0 and store as device_scalar
     */
    /* STEP 1.1(b) Implement the appropriate kernel configuration and
     * for the initial values given, check you have the correct result.
     * Remember to copy result back to host. */

     vector_vector<<< vBlocks, threadsPerBlock >>>(device_vectorR, device_vectorR, device_scalar);
     checkCUDAError("vector_vector<<< vBlocks, threadsPerBlock >>>");

    hipMemcpy(&scalar, device_scalar, scalar_sz, hipMemcpyDeviceToHost);
    checkCUDAError("Memcpy: D2H vector");

     check_vector_vector(vectorR, vectorR, &scalar);
    
    float initial_rs = scalar;
    printf("Initial Rs = %f\n", initial_rs);
    float rsold = initial_rs;
    float beta = 0;
    float alpha = 0;
    float rsnew = 0;

    /*
     * This is the end of the initialisation step
     * We have derived an initial R_0, computed Rs and set P = R_0
     */


    /*
     * This is the start of the main loop
     * We now need to compute alpha, then R_k+1, beta, P_k+1 etc.
     * Once we have computed the value of (R_k+1)s, ie the updated residual, we can stop.
     */   
    int k = 0;
    for (k = 0; abs(rsold) > 1e-5 && k < ARRAY_SIZE; k++){
    
      /*
       * Compute vector Ap_k and store, temporarily, in Pnew
       */
      /* STEP 1.2(b) Use matrix_vector<<<>>>(); */

      hipMemcpy(device_vectorP, vectorP, vector_sz, hipMemcpyHostToDevice);
      checkCUDAError("cudaMemcpy(device_vectorP, vectorP");
      matrix_vector<<<mBlocks,threadsPerBlock>>>(device_matrixA, device_vectorP, device_vectorAP);
      checkCUDAError("matrix_vector<<<mBlocks,threadsPerBlock>>>");

      /*
       * Compute Ap_k dot p_k
       */

      /* STEP 1.1(b) vector_vector<<<>>>(); */

    scalar = 0;
    hipMemcpy(device_scalar, &scalar, scalar_sz, hipMemcpyHostToDevice);
    vector_vector<<<vBlocks, threadsPerBlock>>>(device_vectorP, device_vectorAP, device_scalar);
    hipMemcpy(&scalar, device_scalar, scalar_sz, hipMemcpyDeviceToHost);
    checkCUDAError("vector_vector<<<vBlocks, threadsPerBlock>>>");

      /*
       * Compute Alpha
       */
      alpha = 0;
      alpha = rsold / scalar;

      /*
       * Compute x_k+1 = x_k + alpha p_k
       * Store in Xnew
       */
      /* STEP 1.4(b) Use vector_add_factor<<<>>>(): */

    hipMemcpy(device_vectorX, vectorX, vector_sz, hipMemcpyHostToDevice);
    vector_add_factor<<<vBlocks, threadsPerBlock>>>(device_vectorX, device_vectorP, alpha, device_vectorXnew);
    checkCUDAError("vector_add_factor<<<vBlocks, threadsPerBlock>>>");
    hipMemcpy(vectorXnew,device_vectorXnew,vector_sz,hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy vector_add_factor<<<vBlocks,threadsPerBlock");

      /*
       * Compute r_k+1 = r_k - alpha Ap_k
       * Store in Rnew
       */
      /* STEP 1.4(b) Use vector_add_factor<<<>>>(): */

    hipMemcpy(device_vectorR, vectorR, vector_sz, hipMemcpyHostToDevice);
    vector_add_factor<<<vBlocks, threadsPerBlock>>>(device_vectorR, device_vectorAP, -alpha, device_vectorRnew);
    checkCUDAError("vector_minus_factor<<<vBlocks, threadsPerBlock");
    hipMemcpy(vectorRnew,device_vectorRnew,vector_sz,hipMemcpyDeviceToHost);
    checkCUDAError("Memcpy: vector_minus_factor<<<vBlocks, threadsPerBlock");

      /* Calculate beta = r_k+1 r_k+1 / r_k r_k
       * Recall that we have the denominator r_k r_k as "rsold" */

      scalar = 0;
      /* STEP 1.1(b) vector_vector<<<>>>(); */
    hipMemcpy(device_scalar, &scalar, scalar_sz, hipMemcpyHostToDevice);
    vector_vector<<<vBlocks, threadsPerBlock>>>(device_vectorRnew, device_vectorRnew, device_scalar);
    checkCUDAError("vector_vector<<<vBlocks, threadsPerBlock");

    hipMemcpy(&rsnew, device_scalar, scalar_sz, hipMemcpyDeviceToHost);
    checkCUDAError("Get residual");
    printf("  iteration %d, residual: %f\n", k, rsnew );

      beta = rsnew / rsold;

      /* Compute  p_k+1 = r_k+1 + beta p_k  and store in "Pnew" */

      /* STEP 1.4(b) vector_add_factor<<<>>>(); */

    vector_add_factor<<<vBlocks, threadsPerBlock>>>(device_vectorRnew, device_vectorP, beta, device_vectorPnew);
    checkCUDAError("vector_add_factor<<<vBlocks, threadsPerBlock>>>");
    hipMemcpy(vectorPnew,device_vectorPnew,vector_sz,hipMemcpyDeviceToHost);
    checkCUDAError("hipMemcpy vector_add_factor<<<vBlocks, threadsPerBlock");

      /*
       * Set up for next iteration; copy host vectors.
       */
      rsold = rsnew;
      memcpy(vectorP, vectorPnew, vector_sz);
      memcpy(vectorR, vectorRnew, vector_sz);
      memcpy(vectorX, vectorXnew, vector_sz);
      
    }

    /* STEP 1.5 Recover solution x vector to host */

    /*
     * Free the device memory
     */
    hipFree(device_matrixA);
    hipFree(device_vectorR);
    hipFree(device_vectorB);
    hipFree(device_vectorP);
    hipFree(device_vectorX);
    hipFree(device_vectorRnew);
    hipFree(device_vectorPnew);
    hipFree(device_vectorXnew);
    hipFree(device_vectorAP);
    hipFree(device_scalar);
    
  }


  /*
   * Print the output vector and then free the host memory
   */

  for (i = 0; i < ARRAY_SIZE; i++) {
    printf("%d ", vectorX[i]);
  }
  printf("\n\n");

  free(matrixA);
  free(vectorR);
  free(vectorB);
  free(vectorP);
  free(vectorX);
  free(vectorRnew);
  free(vectorPnew);
  free(vectorXnew);

  return 0;
}


/* Utility function to check for and report CUDA errors */
void checkCUDAError(const char * msg) {

  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
